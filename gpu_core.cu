#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gpu_core.h"
#include "parameters.h"

__device__ float d_L, d_RCUT2;

static float *d_rx, *d_ry, *d_rz;
static float *d_fx, *d_fy, *d_fz;
static float *d_epot, *d_pres;

__device__ float minimum_image_cuda(float cordi, float cell_length)
{
    const float half = 0.5f * cell_length;
    cordi -= cell_length * (cordi > half);
    cordi += cell_length * (cordi <= -half);
    return cordi;
}

__global__ void forces_kernel(
    const float* __restrict__ rx,
    const float* __restrict__ ry,
    const float* __restrict__ rz,
    float* __restrict__ fx,
    float* __restrict__ fy,
    float* __restrict__ fz,
    float* __restrict__ epot,
    float* __restrict__ pres)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    float xi = rx[i];
    float yi = ry[i];
    float zi = rz[i];

    float fxi = 0.0f, fyi = 0.0f, fzi = 0.0f;
    float epot_i = 0.0f, virial = 0.0f;

    for (int j = 0; j < N; j++) {
        if (j == i) continue;

        float rx_ = xi - rx[j];
        float ry_ = yi - ry[j];
        float rz_ = zi - rz[j];

        rx_ = minimum_image_cuda(rx_, d_L);
        ry_ = minimum_image_cuda(ry_, d_L);
        rz_ = minimum_image_cuda(rz_, d_L);

        float rij2 = rx_ * rx_ + ry_ * ry_ + rz_ * rz_;
        if (rij2 <= d_RCUT2) {
            float inv_r2 = 1.0f / rij2;
            float r6 = inv_r2 * inv_r2 * inv_r2;
            float fr = 24.0f * inv_r2 * r6 * (2.0f * r6 - 1.0f);

            fxi += fr * rx_;
            fyi += fr * ry_;
            fzi += fr * rz_;

            epot_i += 4.0f * r6 * (r6 - 1.0f);
            virial += fr * rij2;
        }
    }

    fx[i] = fxi;
    fy[i] = fyi;
    fz[i] = fzi;

    atomicAdd(epot, epot_i * 0.5f); // evitar doble conteo
    atomicAdd(pres, virial * 0.5f);
}

extern "C" void gpu_init() {
    hipMalloc(&d_rx, N * sizeof(float));
    hipMalloc(&d_ry, N * sizeof(float));
    hipMalloc(&d_rz, N * sizeof(float));
    hipMalloc(&d_fx, N * sizeof(float));
    hipMalloc(&d_fy, N * sizeof(float));
    hipMalloc(&d_fz, N * sizeof(float));
    hipMalloc(&d_epot, sizeof(float));
    hipMalloc(&d_pres, sizeof(float));
}

extern "C" void gpu_free() {
    hipFree(d_rx); hipFree(d_ry); hipFree(d_rz);
    hipFree(d_fx); hipFree(d_fy); hipFree(d_fz);
    hipFree(d_epot); hipFree(d_pres);
}

extern "C" void gpu_compute_forces(
    const float* rx, const float* ry, const float* rz,
    float* fx, float* fy, float* fz,
    float* epot, float* pres,
    const float temp, float rho, float V, float L)
{
    float rcut2 = RCUT * RCUT;

    hipMemcpy(d_rx, rx, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ry, ry, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_rz, rz, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_fx, 0, N * sizeof(float));
    hipMemset(d_fy, 0, N * sizeof(float));
    hipMemset(d_fz, 0, N * sizeof(float));
    hipMemset(d_epot, 0, sizeof(float));
    hipMemset(d_pres, 0, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(d_L), &L, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_RCUT2), &rcut2, sizeof(float));

    int blockSize = 128;
    int gridSize = (N + blockSize - 1) / blockSize;

    forces_kernel<<<gridSize, blockSize>>>(
        d_rx, d_ry, d_rz, d_fx, d_fy, d_fz, d_epot, d_pres
    );

    hipMemcpy(fx, d_fx, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(fy, d_fy, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(fz, d_fz, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(epot, d_epot, sizeof(float), hipMemcpyDeviceToHost);

    float pres_vir;
    hipMemcpy(&pres_vir, d_pres, sizeof(float), hipMemcpyDeviceToHost);
    *pres = temp * rho + pres_vir / (3.0f * V);
}
