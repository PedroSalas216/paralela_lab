#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "parameters.h"

__device__ float epot_d, pres_d;

float *d_rx, *d_ry, *d_rz;
float *d_fx, *d_fy, *d_fz;

__global__ void compute_forces_kernel(const float* rx, const float* ry, const float* rz,
                                      float* fx, float* fy, float* fz,
                                      float temp, float rho, float V, float L)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    float xi = rx[i];
    float yi = ry[i];
    float zi = rz[i];

    float fx_i = 0.0f, fy_i = 0.0f, fz_i = 0.0f;
    float epot_local = 0.0f, pres_local = 0.0f;

    for (int j = 0; j < N; j++) {
        if (j == i) continue;

        float dx = xi - rx[j];
        float dy = yi - ry[j];
        float dz = zi - rz[j];

        // PBC
        dx -= L * rintf(dx / L);
        dy -= L * rintf(dy / L);
        dz -= L * rintf(dz / L);

        float r2 = dx*dx + dy*dy + dz*dz;
        if (r2 < RCUT2) {
            float r6 = r2 * r2 * r2;
            float f = 48.0f * (1.0f/r6/r6 - 0.5f/r6) / r2;
            fx_i += f * dx;
            fy_i += f * dy;
            fz_i += f * dz;
            epot_local += 4.0f * (1.0f/r6/r6 - 1.0f/r6);
            pres_local += f * r2; // crude virial contribution
        }
    }
    fx[i] = fx_i;
    fy[i] = fy_i;
    fz[i] = fz_i;

    atomicAdd(&epot_d, epot_local);
    atomicAdd(&pres_d, pres_local / (3.0f * V));
}

extern "C" void gpu_init() {
    hipMalloc(&d_rx, N * sizeof(float));
    hipMalloc(&d_ry, N * sizeof(float));
    hipMalloc(&d_rz, N * sizeof(float));
    hipMalloc(&d_fx, N * sizeof(float));
    hipMalloc(&d_fy, N * sizeof(float));
    hipMalloc(&d_fz, N * sizeof(float));
}

extern "C" void gpu_free() {
    hipFree(d_rx); hipFree(d_ry); hipFree(d_rz);
    hipFree(d_fx); hipFree(d_fy); hipFree(d_fz);
}

extern "C" void gpu_compute_forces(const float* rx, const float* ry, const float* rz,
                                    float temp, float rho, float V, float L) {
    hipMemcpy(d_rx, rx, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ry, ry, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_rz, rz, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemset(d_fx, 0, N * sizeof(float));
    hipMemset(d_fy, 0, N * sizeof(float));
    hipMemset(d_fz, 0, N * sizeof(float));

    hipMemset(&epot_d, 0, sizeof(float));
    hipMemset(&pres_d, 0, sizeof(float));

    int blockSize = 128;
    int numBlocks = (N + blockSize - 1) / blockSize;

    compute_forces_kernel<<<numBlocks, blockSize>>>(d_rx, d_ry, d_rz, d_fx, d_fy, d_fz, temp, rho, V, L);
    hipDeviceSynchronize();
}

extern "C" void gpu_get_forces(float* fx, float* fy, float* fz, float* epot, float* pres) {
    hipMemcpy(fx, d_fx, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(fy, d_fy, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(fz, d_fz, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(epot, &epot_d, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(pres, &pres_d, sizeof(float), hipMemcpyDeviceToHost);
}

